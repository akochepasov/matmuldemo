#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/extrema.h>

#include <hipblas.h>

#include <cutlass/gemm/device/gemm.h>

#include "matmuldemo.h"


__global__ void matmul_kernel1D(int n, float* A, float* B, float* C) {
    float alpha = 1.f, beta = 0.f;
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int i = id / n; int j = id % n;
    // int j = id / n; int i = id % n; // This access pattern 10-100x slower

    C[i * n + j] *= beta;

    for (int k = 0; k < n; k++)
        C[i * n + j] += alpha * A[i * n + k] * B[k * n + j];
}

__global__ void matmul_kernel2D(int n, float *A, float *B, float *C) {
    float alpha = 1.f, beta = 0.f;
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    int col = blockIdx.y * blockDim.y + threadIdx.y;

    if (!(row < n && col < n))
        return;

    float dotProd = 0;
    for (int i = 0; i < n; i++)
        dotProd += A[row * n + i] * B[i * n + col];
    C[row * n + col] = beta * C[row * n + col] + alpha * dotProd;
}

void matmul_cuda1D(int n, int nthreads, float* A, float* B, float* C) {
    thrust::device_vector<float> dvA(A, A + n * n);
    thrust::device_vector<float> dvB(B, B + n * n);
    thrust::device_vector<float> dvC(n * n);

    int nblocks = CEIL_DIV(n, nthreads);

    matmul_kernel1D<<<nblocks * n, nthreads>>>(n,
        thrust::raw_pointer_cast(&dvA[0]),
        thrust::raw_pointer_cast(&dvB[0]),
        thrust::raw_pointer_cast(&dvC[0]));

    thrust::copy(dvC.begin(), dvC.end(), C);
}

void matmul_cuda2D(int n, int nthreads, float* A, float* B, float* C) {
    thrust::device_vector<float> dvA(A, A + n * n);
    thrust::device_vector<float> dvB(B, B + n * n);
    thrust::device_vector<float> dvC(n * n);

    int nblocks = CEIL_DIV(n, nthreads);

    dim3 blksPerGrid(nblocks, nblocks);
    dim3 thrsPerBlock(nthreads, nthreads);

    matmul_kernel2D<<<blksPerGrid, thrsPerBlock>>>(n,
        thrust::raw_pointer_cast(&dvA[0]),
        thrust::raw_pointer_cast(&dvB[0]),
        thrust::raw_pointer_cast(&dvC[0]));

    thrust::copy(dvC.begin(), dvC.end(), C);
}

void matmul_cublas(int n, float* A, float* B, float* C) {
    const float alpha = 1.0, beta = 0.0;

    thrust::device_vector<float> dvA(A, A + n * n);
    thrust::device_vector<float> dvB(B, B + n * n);
    thrust::device_vector<float> dvC(n * n);

    int lda = n, ldb = n, ldc = n;

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n,
        &alpha, dvB.data().get(), ldb, dvA.data().get(), lda,
        &beta, thrust::raw_pointer_cast(&dvC[0]), ldc);

    thrust::copy(dvC.begin(), dvC.end(), C);
    hipblasDestroy(handle);
}

void matmul_cutlass(int n, float *A, float *B, float *C) {
    const float alpha = 1.0, beta = 0.0;

    thrust::device_vector<float> dvA(A, A + n * n);
    thrust::device_vector<float> dvB(B, B + n * n);
    thrust::device_vector<float> dvC(n * n);

    using clMajor = cutlass::layout::ColumnMajor;
    using clGemm = cutlass::gemm::device::Gemm<float,     // Data-type of A matrix
                                                clMajor,  // Layout of A matrix
                                                float,    // Data-type of B matrix
                                                clMajor,  // Layout of B matrix
                                                float,    // Data-type of C matrix
                                                clMajor>; // Layout of C matrix

    float *dA = dvA.data().get();
    float *dB = dvB.data().get();
    float *dC = thrust::raw_pointer_cast(&dvC[0]);

    int lda = n, ldb = n, ldc = n;
    clGemm::Arguments args( {n, n, n},      // Gemm dimensions
                            {dB, ldb},      // Tensor-ref for source matrix B
                            {dA, lda},      // Tensor-ref for source matrix A
                            {dC, ldc},      // Tensor-ref for source matrix B
                            {dC, ldc},      // Tensor-ref for destination matrix D (may be different memory than source C matrix)
                            {alpha, beta}); // Scalars used in the Epilogue

    clGemm()(args);

    thrust::copy(dvC.begin(), dvC.end(), C);
}
